#include "hip/hip_runtime.h"
#include "Application.hpp"
#include "VolumeRayCaster.hpp"

#include <vector>
#include <thread>

#include <GcCore/libData/Configuration.hpp>
#include <GcCore/libData/VolumeConfiguration.hpp>
#include <GcCore/cuda/libPreprocessor/Mipmapper.hpp>
#include <GcCore/cuda/libPreprocessor/BrickProcessor.hpp>
#include <GcCore/cuda/libPreprocessor/BrickProcessorPredicate.hpp>
#include <GcCore/libPreprocessor/Bricker_v2.hpp>
#include <GcCore/libCommon/FileSystem.hpp>
#include <GcCore/libCommon/Logger/Logger.hpp>
#include <GcCore/libMath/Vector.hpp>
#include <GcCore/cuda/libGPUCache/CacheManager.hpp>

namespace tdns
{
namespace app
{
    //---------------------------------------------------------------------------------------------
    bool Application::init(const std::string& cfg_file) const
    {
        tdns::data::Configuration &conf = tdns::data::Configuration::get_instance();

        // Load configuration file
        conf.load<tdns::data::TDNSConfigurationParser>(cfg_file);

        if (!data_folder_check())
            return false;

        //step doing when opening a new volume
        std::string fileName, workingDirectory;
        conf.get_field("VolumeFile", fileName);
        conf.get_field("WorkingDirectory", workingDirectory);
        std::string volumeDirectory = workingDirectory + tdns::common::get_file_base_name(fileName) + "/";
        conf.add_field("VolumeDirectory", volumeDirectory);

        return true;
    }

    //---------------------------------------------------------------------------------------------
    void Application::run(const std::string& cfg_file)
    {
        tdns::data::Configuration &conf = tdns::data::Configuration::get_instance();
        std::string volumeDirectory;
        conf.get_field("VolumeDirectory", volumeDirectory);

        int32_t gpuID;
        CUDA_SAFE_CALL(hipGetDevice(&gpuID));

        tdns::data::MetaData volumeData;

        // Get the brick size in the configuration
        uint32_t brickSize;
        conf.get_field("BrickSize", brickSize);

        // Create or load the multi-resolution bricked representation of the volume to visualize
        std::string bricksDirectory = volumeDirectory + 
            tdns::data::BricksManager::get_brick_folder(tdns::math::Vector3ui(brickSize));
        if (!tdns::common::is_dir(bricksDirectory))
        {
            LOGINFO(10, tdns::common::log_details::Verbosity::INSANE, 
                "Bricks folder does not exist, start preprocessing... [" << bricksDirectory << "].");
            pre_process(volumeData);
        }
        else if (!volumeData.load()) return;

        LOGINFO(10, tdns::common::log_details::Verbosity::INSANE, "Bricks folder found [" << bricksDirectory << "].");

        // Determine the configuration file of the volume(s) to visualize
        std::vector<tdns::data::VolumeConfiguration> volumeConfigurations(1); // only one volume here
        volumeConfigurations[0] = tdns::data::load_volume_configuration(cfg_file);

        // Cache configuration (size) 
        // (here we use only one level of pagination)
        std::vector<tdns::math::Vector3ui> blockSize(1, brickSize);
        std::vector<tdns::math::Vector3ui> cacheSize(1, tdns::math::Vector3ui(25, 25, 25));
        // cache size of (25x25x25) bricks of (16x16x16) voxels of uchar1 bytes = 25x25x25x16x16x16x1 = 61Mo
        // (Must be adapted to bricks size, voxels encoding type AND GPU available memory!).
        // (If it's too large, it will cause an [out of memory] error. If it's too small, the cache will fill up quickly and performance will suffer!)

        tdns::data::CacheConfiguration cacheConfiguration;
        cacheConfiguration.CacheSize = cacheSize;
        cacheConfiguration.BlockSize = blockSize;

        uint32_t numBytes;
        conf.get_field("NumberEncodedBytes", numBytes);

        // Create the GPU Cache Manager and run raycaster
        switch (numBytes) {
            case 1: {
                cacheConfiguration.DataCacheFlags = 1;
                std::unique_ptr<tdns::gpucache::CacheManager<uchar1>> cacheManager;
                cacheManager = tdns::common::create_unique_ptr<tdns::gpucache::CacheManager<uchar1>>(volumeConfigurations[0], cacheConfiguration, gpuID);

                tdns::graphics::display_volume_raycaster(cacheManager.get(), volumeData);
                break;
            }
            case 2: {
                cacheConfiguration.DataCacheFlags = 1;
                std::unique_ptr<tdns::gpucache::CacheManager<ushort1>> cacheManager;
                cacheManager = tdns::common::create_unique_ptr<tdns::gpucache::CacheManager<ushort1>>(volumeConfigurations[0], cacheConfiguration, gpuID);

                tdns::graphics::display_volume_raycaster(cacheManager.get(), volumeData);
                break;
            }
            case 4: {
                cacheConfiguration.DataCacheFlags = 1;
                std::unique_ptr<tdns::gpucache::CacheManager<float1>> cacheManager;
                cacheManager = tdns::common::create_unique_ptr<tdns::gpucache::CacheManager<float1>>(volumeConfigurations[0], cacheConfiguration, gpuID);

                tdns::graphics::display_volume_raycaster(cacheManager.get(), volumeData);
                break;
            }
            default:
                LOGFATAL(10, "Invalid value for 'NumberEncodedBytes'");
        }
    }

    //---------------------------------------------------------------------------------------------
    bool Application::data_folder_check() const
    {
        if (tdns::common::is_dir("data")) return true;
        
        if (tdns::common::is_file("data"))
        {
            LOGFATAL(10, "Data already exist next to the binary and is not a folder.");
            return false;
        }

        LOGTRACE(10, tdns::common::log_details::Verbosity::INSANE, "Create the folder \"data\" next to the binary.");
        tdns::common::create_folder("data");
        return true;
    }

    //---------------------------------------------------------------------------------------------
    void Application::pre_process(tdns::data::MetaData &volumeData) const
    {
        tdns::data::Configuration &conf = tdns::data::Configuration::get_instance();
        std::cout << "Start pre-processing (see log file) ..." << std::endl;

        uint32_t numBytes;
        conf.get_field("NumberEncodedBytes", numBytes);

        // Mipmapping
        tdns::preprocessor::Mipmapper mipmapper;
        mipmapper.process(volumeData);

        // Bricking
        std::vector<tdns::math::Vector3ui> levels = volumeData.get_initial_levels();
        pre_process_bricking(volumeData, levels);

        // PROCESS EMPTY BRICKS AND VOLUME HISTOGRAM
        // ========================= UCHAR1 =========================
        uint32_t *d_threshold;
        uint32_t threshold = 0;
        CUDA_SAFE_CALL(hipMalloc(&d_threshold, sizeof(uint32_t)));
        CUDA_SAFE_CALL(hipMemcpy(d_threshold, &threshold, sizeof(uint32_t), hipMemcpyHostToDevice));

        switch (numBytes) {
            case 1: {
                tdns::preprocessor::BrickProcessor<uchar1> brickProcessor(volumeData);
                brickProcessor.process_histo();
                brickProcessor.process_empty<tdns::preprocessor::DefaultBrickProcessorPredicate>(d_threshold);
                break;
            }
            case 2: {
                tdns::preprocessor::BrickProcessor<ushort1> brickProcessor(volumeData);
                brickProcessor.process_histo();
                brickProcessor.process_empty<tdns::preprocessor::BrickProcessor16BitsPredicate>(d_threshold);
                break;
            }
            case 4: {
                tdns::preprocessor::BrickProcessor<float1> brickProcessor(volumeData);
                // Not available for float1 apparently
                //brickProcessor.process_histo();
                brickProcessor.process_empty<tdns::preprocessor::BrickProcessorF32Predicate>(d_threshold);
                break;
            }
            default:
                LOGFATAL(10, "Invalid value for 'NumberEncodedBytes'");
        }

        volumeData.write_bricks_xml();
    }

    //---------------------------------------------------------------------------------------------
    void Application::pre_process_mipmapping(tdns::data::MetaData &volumeData) const
    {
        tdns::preprocessor::MipmappingConfiguration configuration;
        tdns::data::Configuration &conf = tdns::data::Configuration::get_instance();

        //Volumedirectory
        conf.get_field("VolumeDirectory", configuration.volumeDirectory);
        //volume file name
        conf.get_field("VolumeFile", configuration.volumeFileName);
        //Outputdirectory
        conf.get_field("VolumeDirectory", configuration.outputDirectory);
        //Level dimensions
        conf.get_field("size_X", configuration.levelDimension[0]);
        conf.get_field("size_Y", configuration.levelDimension[1]);
        conf.get_field("size_Z", configuration.levelDimension[2]);
        //Down sampling ratios
        conf.get_field("downScale_X", configuration.downScaleRatio[0]);
        conf.get_field("downScale_Y", configuration.downScaleRatio[1]);
        conf.get_field("downScale_Z", configuration.downScaleRatio[2]);
        //brick Size
        uint32_t brickSize;
        conf.get_field("BrickSize", brickSize);
        configuration.brickSize = tdns::math::Vector3ui(brickSize);
        //EncodedByte
        conf.get_field("NumberEncodedBytes", configuration.encodedBytes);
        //Number of channels
        conf.get_field("NumberChannels", configuration.numberChannels);

        tdns::preprocessor::process_mipmapping(configuration);
        tdns::preprocessor::fill_metaData(volumeData.get_initial_levels(), volumeData.nb_levels(), configuration.levelDimension, configuration.downScaleRatio);
    }

    //---------------------------------------------------------------------------------------------
    void Application::pre_process_bricking(tdns::data::MetaData &volumeData, const std::vector<tdns::math::Vector3ui> &levels) const
    {
        tdns::preprocessor::BrickingConfiguration configuration;
        tdns::data::Configuration &conf = tdns::data::Configuration::get_instance();
        //brick Size
        uint32_t brickSize;
        conf.get_field("BrickSize", brickSize);
        configuration.brickSize = tdns::math::Vector3ui(brickSize);
        //EncodedByte
        conf.get_field("NumberEncodedBytes", configuration.encodedBytes);
        //covering
        conf.get_field("VoxelCovering", configuration.covering);
        //Volumedirectory
        conf.get_field("VolumeDirectory", configuration.volumeDirectory);
        //volume file name
        conf.get_field("VolumeFile", configuration.volumeFileName);
        //Outputdirectory
        conf.get_field("VolumeDirectory", configuration.outputDirectory);
        //compression ?
        configuration.compression = true;
        //big brick size
        tdns::math::Vector3ui bigBrickSize;
        conf.get_field("BigBrickSizeX", bigBrickSize[0]);
        conf.get_field("BigBrickSizeY", bigBrickSize[1]);
        conf.get_field("BigBrickSizeZ", bigBrickSize[2]);
        configuration.bigBrickSize = bigBrickSize;

        //fill volumeData.
        tdns::preprocessor::init_meta_data(volumeData, configuration, levels);

        std::vector<std::thread> threads(levels.size());
        for (uint32_t i = 0; i < threads.size(); ++i)
        {
            threads[i] = std::thread([&, i, configuration]() mutable
            {
                configuration.level = i;
                configuration.levelDimensionX = levels[i][0];
                configuration.levelDimensionY = levels[i][1];
                configuration.levelDimensionZ = levels[i][2];

                configuration.startX = configuration.startY = configuration.startZ = 0;
                configuration.endX = configuration.levelDimensionX = levels[i][0];
                configuration.endY = configuration.levelDimensionY = levels[i][1];
                configuration.endZ = configuration.levelDimensionZ = levels[i][2];

                tdns::preprocessor::process_bricking(configuration);
            });
        }

        for (size_t i = 0; i < threads.size(); ++i)
            if (threads[i].joinable())
                threads[i].join();
    }
} // namespace app
} // namespace tdns
